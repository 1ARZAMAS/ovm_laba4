
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <chrono>

#define IDX2C(i,j,n) (((i)*(n))+(j))  // макрос для индексации

__global__ void dgemm_kernel(int n, const double* A, const double* B, double* C, double alpha, double beta) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // строка
    int col = blockIdx.x * blockDim.x + threadIdx.x; // столбец

    if (row < n && col < n) {
        double sum = 0.0;
        for (int k = 0; k < n; ++k) {
            sum += A[IDX2C(row, k, n)] * B[IDX2C(k, col, n)];
        }
        C[IDX2C(row, col, n)] = alpha * sum + beta * C[IDX2C(row, col, n)];
    }
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cout << "Использование: " << argv[0] << " <размер матрицы> <потоки на блок>"<< std::endl;
        return 1;
    }

    int n = atoi(argv[1]);
    int threadsPerBlock = (argc >= 3) ? atoi(argv[2]) : 16;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    if (n <= 0) {
        std::cerr << "Ошибка: размер должен быть положительным числом" << std::endl;
        return 1;
    }

    size_t bytes = n * n * sizeof(double);
    std::vector<double> h_A(n * n), h_B(n * n), h_C(n * n, 0.0);

    srand(static_cast<unsigned>(time(nullptr)));
    for (int i = 0; i < n * n; ++i) {
        h_A[i] = static_cast<double>(rand()) / RAND_MAX;
        h_B[i] = static_cast<double>(rand()) / RAND_MAX;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C.data(), bytes, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (n + 15) / 16);

    auto begin = std::chrono::steady_clock::now();
    dgemm_kernel<<<blocks, threads>>>(n, d_A, d_B, d_C, 1.0, 0.0);
    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();

    hipMemcpy(h_C.data(), d_C, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    auto elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end - begin);
    std::cout << "Time: " << elapsed.count() << " sec" << std::endl;
    return 0;
}
